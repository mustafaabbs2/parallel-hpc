
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloFromGPU(void) { printf("Hello World from GPU! \n"); }

int main()
{
    printf("Hello World from GPU! \n");
    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();
    return 0;
}